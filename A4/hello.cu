
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y, int* res)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  printf("%d\n", i);
  res[i] = 1;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 100;
  float *x, *y, *d_x, *d_y;
  
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));


  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  int *result = new int[N];
  memset(result, 0, N*sizeof(int));
  int *d_result;
  hipMalloc(&d_result, N*sizeof(float));
  hipMemcpy(d_result, result, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y, d_result);

  hipMemcpy(result, d_result, N*sizeof(int), hipMemcpyDeviceToHost);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  for(int i=0; i<N; i++){
    printf("%d\n", result[i]);
  }

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}