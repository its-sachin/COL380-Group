
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <chrono>
// using namespace std;

void readImage(int* &img, int &m, int &n, std::string fileName, int *&prefixSum, bool isData = false){

    std::ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*4];

    if(isData){
        prefixSum = new int[m*n];
    }

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            int sum = 0;
            for (int k = 0; k < 3; k++){
                file>>img[(i*n+j)*4+k];
                sum+=img[(i*n+j)*4+k];
            }
            img[(i*n+j)*4+3] = sum/3;  
            if(isData){
                prefixSum[i*n+j] = sum/3;
                if(i == 0){
                    if(j!=0){
                        prefixSum[j] += prefixSum[j-1];
                    }
                }
                else{
                    if(j==0)
                        prefixSum[i*n+j] += prefixSum[(i-1)*n];
                    else
                        prefixSum[i*n+j] += prefixSum[(i-1)*n+j] + prefixSum[i*n + j-1] - prefixSum[(i-1)*n+j-1];
                }
            } 
        }
    }
    file.close();
}

__device__
int d_floor(float x){
    return floor(x);
}

__device__
int d_ceil(float x){
    return ceil(x);
}

__device__
int d_round(float x){
    return llrint(x);
}

__device__
float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){
    float xx = a + i*cos(theta) - j*sin(theta);
    float yy = b + i*sin(theta) + j*cos(theta);
    float x = xx - d_floor(xx);
    float y = yy - d_floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[((d_floor(xx))*N + (d_floor(yy)))*4 + ind];
    int z01 = dataImg[((d_floor(xx))*N + (d_ceil(yy)))*4 + ind];
    int z10 = dataImg[((d_ceil(xx))*N + (d_floor(yy)))*4 + ind];
    int z11 = dataImg[((d_ceil(xx))*N + (d_ceil(yy)))*4 + ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

__global__
void checkGeneral(int * dataImg, int * queryImg, int * prefixSum, int M, int N, int m, int n, int queryAvg, double th1, double th2, float pi, float* result){

    int a,b;
    int absi = blockIdx.x*256 + threadIdx.x;
    a = absi/N;
    b = absi%N;

    int angles[3] = {45,0,-45};
    // printf("At start bid: %d tid: %d\n", blockIdx.x, threadIdx.x);
    // if(absi > 20000)
    // printf("abs: %d a: %d b: %d\n",absi, a, b);

    for(int t=0; t<3; t++){
        float theta = angles[t]*pi/180;

        int a1 = a - n*sin(theta);
        int b1 = b;
        int b2 = b + n*cos(theta) + m*sin(theta);
        int a2 = a + m*cos(theta);
        int sum = prefixSum[a1*N + b1] + prefixSum[a2*N + b2] - prefixSum[a1*N + b2] - prefixSum[a2*N + b1];

        printf("a: %d b: %d val: %d\n", a, b, abs(queryAvg-sum));
        

        if(abs(queryAvg-sum)<=th2){
            double sum = 0;
            for (int i = 0; i<m; i++){
                for (int j = 0; j<n; j++){
                    for (int r = 0; r < 3; r++){
                        sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[(i*n+j)*4+r],2)/(m*n*3);
                    }
                }
            }
            // cout << "   -> " <<sqrt(sum) << endl;
            // printf("    -> %f\n",sqrt(sum));
            float sq = sqrt(sum);
            if(sq<=th1){
                int ansx = M-d_round(a + m*cos(theta) );
                int ansy = d_round(b + m*sin(theta) );
                // printf("IRes: %d %d %d %f\n",ansx,ansy,t,sq);
                result[ansx*N*3 + ansy*3 + t] = sq;
                // result[(ansx*N + ansy)*3 = ansx;
                return;
            }
        }
    }
}

int getAvg(int* &queryImg, int m, int n){
    int queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            queryAvg+=queryImg[(i*n+j)*4+3];
        }
    }
    return queryAvg;
}

class container{

    public:
    int x,y,angle;

    container(int a,int b,int c){
        x = a;
        y = b;
        angle = c;
    }
};


int main(int argc, char** argv)
{
    auto start = std::chrono::high_resolution_clock::now();

    std::string dataImgPath = argv[1];
    std::string queryImgPath = argv[2];
    double th1 = std::stod(argv[3]);
    double th2 = std::stod(argv[4]);
    int maxN = std::stoi(argv[5]);

    int M,N,m,n;

    int *dataImg;
    int *queryImg;
    int *dataPrefix;

    readImage(dataImg,M,N,dataImgPath,dataPrefix,true);
    readImage(queryImg,m,n,queryImgPath,dataPrefix);

    th2*=m*n;
    
    int *d_dataImg;
    int *d_queryImg;
    int *d_dataPrefix;
    
   
    hipMalloc(&d_dataImg, (M*N*4)*sizeof(int));
    hipMalloc(&d_queryImg, (m*n*4)*sizeof(int));
    hipMalloc(&d_dataPrefix, (M*N)*sizeof(int));

    hipMemcpy(d_dataImg, dataImg, (M*N*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dataPrefix, dataPrefix, (M*N)*sizeof(int), hipMemcpyHostToDevice);

    float *result = new float[N*M*3];
    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<3; k++){
                result[i*N*3+j*3+k]=-1;
            }
        }
    }
    float *d_result;
    hipMalloc(&d_result, M*N*3*sizeof(float));
    hipMemcpy(d_result, result, M*N*3*sizeof(float), hipMemcpyHostToDevice);

    int queryAvg = getAvg(queryImg, m,n);
    // checkGeneral(dataImg, queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180);

    auto mid = std::chrono::high_resolution_clock::now();

    std::cout << "Pre processing Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(mid - start).count() << " ms" << std::endl;

    checkGeneral<<<(N*M+255)/256, 256>>>(d_dataImg, d_queryImg, d_dataPrefix, M,N,m,n,queryAvg,th1,th2,M_PI,d_result);

    hipMemcpy(result, d_result, M*N*3*sizeof(float), hipMemcpyDeviceToHost);

    std::priority_queue <std::pair<float, container*> > pq;
    int angles[3] = {45,0,-45};
    
    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<3; k++){
                if(result[i*N*3+j*3+k]!=-1){
                    printf("%d %d %d %f\n",i,j,k,result[i*N*3+j*3+k]);
                    container* c = new container(i,j,angles[k]);
                    pq.push({result[i*N*3+j*3+k], c});
                }
            }
        }
    }

    std::cout << std::endl;
    for(int i=0; i<maxN && pq.size() > 0; i++){
        std::pair<float, container*> p = pq.top();
        pq.pop();
        printf("Res[%d]: %d %d %d %f\n",i,p.second->x,p.second->y,p.second->angle,p.first);
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Computation Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - mid).count() << " ms" << std::endl;
    std::cout << "Total Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    hipFree(d_dataImg);
    hipFree(d_queryImg);
    hipFree(d_result);
    delete(dataImg);
    delete(result);
    delete(queryImg);
}
