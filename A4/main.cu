
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void readImage(int* &img, int &m, int &n, string fileName){

    ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*4];

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            int sum = 0;
            for (int k = 0; k < 3; k++){
                file>>img[(i*n+j)*4+k];
                sum+=img[(i*n+j)*4+k];
            }
            img[(i*n+j)*4+3] = sum/3;   
        }
    }
    file.close();
}


__device__
float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){
    float xx = a + i*cos(theta) - j*sin(theta);
    float yy = b + i*sin(theta) + j*cos(theta);
    float x = xx - floor(xx);
    float y = yy - floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[(((int)floor(xx))*N + ((int)floor(yy)))*4 + ind];
    int z01 = dataImg[(((int)floor(xx))*N + ((int)ceil(yy)))*4 + ind];
    int z10 = dataImg[(((int)ceil(xx))*N + ((int)floor(yy)))*4 + ind];
    int z11 = dataImg[(((int)ceil(xx))*N + ((int)ceil(yy)))*4 + ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

__global__
void checkGeneral(int * &dataImg, int * &queryImg, int M, int N, int m, int n, int queryAvg, double th1, double th2, float theta){

    int a,b;
    int absi = blockIdx.x*256 + threadIdx.x;
    a = absi%M;
    b = absi%N;

    //printf("blockdin: %d\n", blockIdx.x);
    // if(absi > 20000)
    // printf("abs: %d a: %d b: %d\n",absi, a, b);
    float sum = 0;
    //printf("Before interpol a: %d b: %d %f\n", a, b, abs(queryAvg-sum)/(m*n));
    for(int i =0; i<m; i++){
        for(int j=0; j<n; j++){
            sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
        }
    }


    // cout << "a: " << a << " b: " << b << " " << abs(queryAvg-sum)/(m*n) << endl;
    printf("After interpol a: %d b: %d %f\n", a, b, abs(queryAvg-sum)/(m*n));


    if(abs(queryAvg-sum)<=th2){
        double sum = 0;
        for (int i = 0; i<m; i++){
            for (int j = 0; j<n; j++){
                for (int r = 0; r < 3; r++){
                    sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[(i*n+j)*4+r],2)/(m*n*3);
                }
            }
        }
        // cout << "   -> " <<sqrt(sum) << endl;
        printf("    -> %f\n",sqrt(sum));
        if(sqrt(sum)<=th1){
            int ansx = M-std::round(a + m*cos(theta) );
            int ansy = std::round(b + n*sin(theta) );
            int anst = (int)(theta*180/M_PI);
            printf("res = %d %d %d\n",ansx,ansy,anst);
            return;
        }
    }
}

int getAvg(int* &queryImg, int m, int n){
    int queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            queryAvg+=queryImg[(i*n+j)*4+3];
        }
    }
    return queryAvg;
}


int main(int argc, char** argv)
{
    string dataImgPath = argv[1];
    string queryImgPath = argv[2];
    double th1 = stod(argv[3]);
    double th2 = stod(argv[4]);
    int maxN = stoi(argv[5]);

    int M,N,m,n;

    int *dataImg;
    int *queryImg;
    
    readImage(dataImg,M,N,dataImgPath);
    readImage(queryImg,m,n,queryImgPath);

    th2*=m*n;
    
    int *d_dataImg;
    int *d_queryImg;
   
    hipMalloc(&d_dataImg, (M*N*4)*sizeof(int));
    hipMalloc(&d_queryImg, (m*n*4)*sizeof(int));

    hipMemcpy(d_dataImg, dataImg, (M*N*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*4)*sizeof(int), hipMemcpyHostToDevice);

    int queryAvg = getAvg(queryImg, m,n);
    // checkGeneral(dataImg, queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180);

    checkGeneral<<<(N*M+255)/256, 256>>>(d_dataImg, d_queryImg, M,N,m,n,queryAvg,th1,th2,0*M_PI/180);

    hipFree(d_dataImg);
    hipFree(d_queryImg);
    delete(dataImg);
    delete(queryImg);
}
