
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
// using namespace std;

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void readImage(int* &img, int &m, int &n, std::string fileName){

    std::ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*4];

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            int sum = 0;
            for (int k = 0; k < 3; k++){
                file>>img[(i*n+j)*4+k];
                sum+=img[(i*n+j)*4+k];
            }
            img[(i*n+j)*4+3] = sum/3;   
        }
    }
    file.close();
}

__device__
int d_floor(float x){
    return floor(x);
}

__device__
int d_ceil(float x){
    return ceil(x);
}

__device__
int d_round(float x){
    return llrint(x);
}

__device__
float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){
    float xx = a + i*cos(theta) - j*sin(theta);
    float yy = b + i*sin(theta) + j*cos(theta);
    float x = xx - d_floor(xx);
    float y = yy - d_floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[((d_floor(xx))*N + (d_floor(yy)))*4 + ind];
    int z01 = dataImg[((d_floor(xx))*N + (d_ceil(yy)))*4 + ind];
    int z10 = dataImg[((d_ceil(xx))*N + (d_floor(yy)))*4 + ind];
    int z11 = dataImg[((d_ceil(xx))*N + (d_ceil(yy)))*4 + ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

__global__
void checkGeneral(int * dataImg, int * queryImg, int M, int N, int m, int n, int queryAvg, double th1, double th2, float theta, int* result){

    int a,b;
    int absi = blockIdx.x*256 + threadIdx.x;
    a = absi/N;
    b = absi%N;

    // printf("At start bid: %d tid: %d\n", blockIdx.x, threadIdx.x);

    result[absi] = 1;
    // if(absi > 20000)
    // printf("abs: %d a: %d b: %d\n",absi, a, b);
    float sum = 0;    

    // printf("Before interpol bid: %d tid: %d\n", blockIdx.x, threadIdx.x);

    for(int i =0; i<m; i++){
        for(int j=0; j<n; j++){
            sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
        }
    }


    // cout << "a: " << a << " b: " << b << " " << abs(queryAvg-sum)/(m*n) << endl;
    // printf("After interpol bid: %d tid: %d\n", blockIdx.x, threadIdx.x);


    if(abs(queryAvg-sum)<=th2){
        double sum = 0;
        for (int i = 0; i<m; i++){
            for (int j = 0; j<n; j++){
                for (int r = 0; r < 3; r++){
                    sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[(i*n+j)*4+r],2)/(m*n*3);
                }
            }
        }
        // cout << "   -> " <<sqrt(sum) << endl;
        // printf("    -> %f\n",sqrt(sum));
        if(sqrt(sum)<=th1){
            int ansx = M-d_round(a + m*cos(theta) );
            int ansy = d_round(b + n*sin(theta) );
            int anst = (int)(theta*180/M_PI);
            printf("res = %d %d %d\n",ansx,ansy,anst);
            return;
        }
    }
}

int getAvg(int* &queryImg, int m, int n){
    int queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            queryAvg+=queryImg[(i*n+j)*4+3];
        }
    }
    return queryAvg;
}


int main(int argc, char** argv)
{
    std::string dataImgPath = argv[1];
    std::string queryImgPath = argv[2];
    double th1 = std::stod(argv[3]);
    double th2 = std::stod(argv[4]);
    int maxN = std::stoi(argv[5]);

    int M,N,m,n;

    int *dataImg;
    int *queryImg;
    
    readImage(dataImg,M,N,dataImgPath);
    readImage(queryImg,m,n,queryImgPath);

    th2*=m*n;
    
    int *d_dataImg;
    int *d_queryImg;
    
   
    hipMalloc(&d_dataImg, (M*N*4)*sizeof(int));
    hipMalloc(&d_queryImg, (m*n*4)*sizeof(int));

    hipMemcpy(d_dataImg, dataImg, (M*N*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*4)*sizeof(int), hipMemcpyHostToDevice);

    int *result = new int[N*M];
    memset(result, 0, M*N*sizeof(int));
    int *d_result;
    hipMalloc(&d_result, M*N*sizeof(float));
    hipMemcpy(d_result, result, M*N*sizeof(float), hipMemcpyHostToDevice);

    int queryAvg = getAvg(queryImg, m,n);
    // checkGeneral(dataImg, queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180);

    checkGeneral<<<(N*M+255)/256, 256>>>(d_dataImg, d_queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180,d_result);

    hipMemcpy(result, d_result, M*N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            if(result[i*N+j]==1){
                printf("%d %d\n",i,j);
            }
        }
    }

    hipFree(d_dataImg);
    hipFree(d_queryImg);
    delete(dataImg);
    delete(queryImg);
}
