#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <chrono>
// using namespace std;

void readImage(int* &img, int &m, int &n, std::string fileName){

    std::ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*3];

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            int sum = 0;
            for (int k = 0; k < 3; k++){
                file>>img[(i*n+j)*3+k];
                sum+=img[(i*n+j)*3+k];
            }
        }
    }
    file.close();
}

void readImage(int* &img, int &m, int &n, std::string fileName, float* &prefix){

    std::ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*3];
    prefix = new float[m*n];

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            float sum = 0;
            for (int k = 0; k < 3; k++){
                file>>img[(i*n+j)*3+k];
                sum+=img[(i*n+j)*3+k];
            }
            prefix[i*n +j]=sum/3;
        }
    }
    file.close();
}

__device__
int d_floor(float x){
    return floor(x);
}

__device__
int d_ceil(float x){
    return ceil(x);
}

__device__
int d_round(float x){
    return llrint(x);
}
__device__
float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){

    float xx = a - i*cos(theta) - j*sin(theta);
    float yy = b - i*sin(theta) + j*cos(theta);
    float x = xx - d_floor(xx);
    float y = yy - d_floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[((d_floor(xx))*N + (d_floor(yy)))*3 + ind];
    int z01 = dataImg[((d_floor(xx))*N + (d_ceil(yy)))*3 + ind];
    int z10 = dataImg[((d_ceil(xx))*N + (d_floor(yy)))*3 + ind];
    int z11 = dataImg[((d_ceil(xx))*N + (d_ceil(yy)))*3 + ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

__global__
void checkGeneral(int * dataImg, int * queryImg, float * prefix, int M, int N, int m, int n, float queryAvg, double th1, double th2, float pi, float* result){

    int a,b;
    int absi = blockIdx.x*256 + threadIdx.x;
    a = absi/N;
    b = absi%N;

    int angles[3] = {45,0,-45};
    int t = threadIdx.y;

    if(a >= M or a < 0 )return ;
    // printf("At start bid: %d tid: %d\n", blockIdx.x, threadIdx.x);
    // if(absi > 20000)
    // printf("abs: %d a: %d b: %d\n",absi, a, b);

    // for(int t=0; t<3; t++){
        // float sum = 0;    
        // float theta = angles[t]*pi/180;

        // // printf("Before interpol bid: %d tid: %d\n", blockIdx.x, threadIdx.x);

        // for(int i =0; i<m; i++){
        //     for(int j=0; j<n; j++){
        //         sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
        //     }
        // }

        
        float theta = angles[t]*pi/180;
        

        // if(M-a-1 == 840 and b == 900)
        // printf("before a: %d b: %d theta: %f th2: %f, queryAvg: %f\n", a, b, theta,th2,queryAvg);

        int a1,b1,a2,b2;

        if(theta < 0){
            a1 = a + m*sin(theta);
            b2 = b + m*cos(theta) - n*sin(theta);
            a2 = a + n*cos(theta);
            b1 = b;
        }

        else{
            a2 = a;
            b1 = b - m*sin(theta) ;
            b2 = b + n*cos(theta);
            a1 = a - n*cos(theta) - m*sin(theta) ;
        }

        int denom = abs((a2-a1)*(b2-b1));

        a1 = max(min(a1,M-1),0);
        b2 = max(min(b2,N-1),0);
        b1 = max(min(b1,N-1),0);
        a2 = max(min(a2,M-1),0);
    
        float sum = (prefix[a1*N + b1] + prefix[a2*N + b2] - prefix[a2*N + b1] - prefix[a1*N + b2])/denom;

        // //printf("p: %d q: %d r: %d , s: %d \n", p, q, r, s);
        // //printf("a1: %d b1: %d a2: %d b2: %d  P: %d Q: %d R: %d S: %d theta: %f \n ", a1, b1,a2,b2 ,a1*N+b1, a2*N+b2, a1*N+b2, a2*N+b1 ,theta);
        // printf("a1: %d b1: %d a2: %d b2: %d  P: %d Q: %d R: %d S: %d theta: %f \n ", a1, b1,a2,b2 ,p,q,r,s,theta);
        // //printf("a: %d b: %d val: %d\n", a, b, abs(queryAvg-sum));
        // //int sum = prefixSum[a1*N + b1] + prefixSum[a2*N + b2] - prefixSum[a1*N + b2] - prefixSum[a2*N + b1];

        // if(M-a-1 == 840 and b == 900){
        //     printf("(a: %d b: %d angle : %d), (sum: %f) , (queryAvg: %f) , (absDiff: %f), (a1 %d, b1 %d, a2 %d, b2 %d), (p11 %f, p22 %f, p21 %f, p12 %f) \n", a, b,angles[t],sum,queryAvg,abs(queryAvg-sum),a1,b,a2,b2,prefix[a1*N + b],prefix[a2*N + b2],prefix[a2*N + b],prefix[a1*N + b2]);
        // }

        if(abs(queryAvg-sum)<=th2){
            double sum = 0;
            for (int i = 0; i<m; i++){
                for (int j = 0; j<n; j++){
                    for (int r = 0; r < 3; r++){
                        sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[((m-i-1)*n+j)*3+r],2)/(m*n*3);
                    }
                }
            }
            
            
            // cout << "   -> " <<sqrt(sum) << endl;
            // printf("    -> %f\n",sqrt(sum));
            float sq = sqrt(sum);
            if(M-a-1 == 840 and b == 900){
                printf("sqrt %f",sq);
            }
            if(sq<=th1){
                int ansx =  M- a-1 ;
                int ansy =  b ;
                // printf("IRes: %d %d %d %f\n",ansx,ansy,t,sq);
                result[ansx*N*3 + ansy*3 + t] = sq;
                // result[(ansx*N + ansy)*3 = ansx;
                return;
            // }
        }
    }
}

float getAvg(int* &queryImg, int m, int n){
    float queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            float sum = 0;
            for(int k=0; k<3; k++)
                sum += queryImg[(i*n+j)*3+k];
            queryAvg += sum/3;
        }
    }
    return queryAvg/(m*n);
}

__global__
void rowsum(float* arr, int m, int n){
    int rownum = blockIdx.x*256 + threadIdx.x;
    if(rownum>=m)return;
    for(int i=1; i<n;i++){
        arr[rownum*n+i]+=arr[rownum*n+i-1];
    }
}

__global__
void colsum(float* arr, int m, int n){
    int colnum = blockIdx.x*256 + threadIdx.x;
    if(colnum>=n)return;
    for(int i=1; i<m;i++){
        arr[i*n+colnum]+=arr[(i-1)*n+colnum];
    }
}


void setSum(int *a, float *&sum, int m, int n){

    sum = new float[m*n];
    sum[0] = (a[0]+a[1]+a[2])/3;
 
    for (int i=1; i<n; i++){
        float s = 0;
        for(int k=0; k<3; k++)s+=a[i*3+k];
        sum[i] = sum[i-1] + s/3; 
    }
    for (int i=1; i<m; i++){
        float s = 0;
        for(int k=0; k<3; k++)s+=a[i*n*3+k];
        sum[i*n] = sum[(i-1)*n] + s/3;
    }
 
    for (int i=1; i<m; i++){
        for (int j=1; j<n; j++){
            float s = 0;
            for(int k=0; k<3; k++)s+=a[i*n*3+j*3+k];
            sum[i*n+j] = sum[(i - 1)*n+j] + sum[i*n + j - 1] - sum[(i - 1)*n + j - 1] + s/3;
        }
    }
}

class container{

    public:
    int x,y,angle;

    container(int a,int b,int c){
        x = a;
        y = b;
        angle = c;
    }
};




int main(int argc, char** argv)
{
    auto start = std::chrono::high_resolution_clock::now();

    std::string dataImgPath = argv[1];
    std::string queryImgPath = argv[2];
    double th1 = std::stod(argv[3]);
    double th2 = std::stod(argv[4]);
    int maxN = std::stoi(argv[5]);

    int M,N,m,n;

    int *dataImg;
    int *queryImg;
    float *dataPrefix;

    // std::cout << "1" << std::endl;

    readImage(dataImg,M,N,dataImgPath,dataPrefix);
    // std::cout << "1.5" << std::endl;
    readImage(queryImg,m,n,queryImgPath);
    // std::cout << "1.75" << std::endl;

    
    int *d_dataImg;
    int *d_queryImg;
     
    hipMalloc(&d_dataImg, (M*N*3)*sizeof(int));
    hipMalloc(&d_queryImg, (m*n*3)*sizeof(int));

    hipMemcpy(d_dataImg, dataImg, (M*N*3)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*3)*sizeof(int), hipMemcpyHostToDevice);

    float *d_dataPrefix;
    hipMalloc(&d_dataPrefix, (M*N)*sizeof(float));
    hipMemcpy(d_dataPrefix, dataPrefix, (M*N)*sizeof(float), hipMemcpyHostToDevice);

    rowsum<<<(M+255)/256,256>>>(d_dataPrefix,M,N);
    hipMemcpy(dataPrefix, d_dataPrefix, (M*N)*sizeof(float), hipMemcpyDeviceToHost);
    colsum<<<(N+255)/256,256>>>(d_dataPrefix,M,N);

    hipMemcpy(dataPrefix, d_dataPrefix, (M*N)*sizeof(float), hipMemcpyDeviceToHost);

    float* temp;
    setSum(dataImg,temp,M,N);

    // hipMemcpy(d_dataPrefix, temp,(M*N)*sizeof(float), hipMemcpyHostToDevice);

    // for(int i=0; i<M;i++){
    //     for(int j=0;j<N;j++)
    //         std::cout<<"("<<temp[i*N+j]<<" " <<dataPrefix[i*N+j] << ")";
    //     std::cout<<std::endl;
    // }

    float *result = new float[N*M*3];
    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<3; k++){
                result[i*N*3+j*3+k]=-1;
            }
        }
    }
    float *d_result;
    hipMalloc(&d_result, M*N*3*sizeof(float));
    hipMemcpy(d_result, result, M*N*3*sizeof(float), hipMemcpyHostToDevice);

    float queryAvg = getAvg(queryImg, m,n);
    // checkGeneral(dataImg, queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180);

    auto mid = std::chrono::high_resolution_clock::now();

    std::cout << "Pre processing Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(mid - start).count() << " ms" << std::endl;

    checkGeneral<<<(N*M+255)/256, {256,3,1}>>>(d_dataImg, d_queryImg, d_dataPrefix, M,N,m,n,queryAvg,th1,th2,M_PI,d_result);

    hipMemcpy(result, d_result, M*N*3*sizeof(float), hipMemcpyDeviceToHost);

    std::priority_queue <std::pair<float, container*> > pq;
    int angles[3] = {45,0,-45};
    
    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<3; k++){
                if(result[i*N*3+j*3+k]!=-1){
                    // printf("%d %d %d %f\n",i,j,k,result[i*N*3+j*3+k]);
                    container* c = new container(i,j,angles[k]);
                    pq.push({result[i*N*3+j*3+k], c});
                }
                if(pq.size()>maxN)pq.pop();
            }
        }
    }
    //std::cout<<"pq size: "<<pq.size()<<std::endl;
    int pqSize = pq.size();
    std::vector<std::pair<float, container*>> vecRes;
    for(int i = 0;i<pqSize;i++){
        vecRes.push_back(pq.top());
        pq.pop();
    }
    std::ofstream outfile("output.txt");
    std::cout << std::endl;
    reverse(vecRes.begin(), vecRes.end());
    for(int i = 0;i<vecRes.size();i++){
        outfile << vecRes[i].second->x << " " << vecRes[i].second->y << " " << vecRes[i].second->angle << std::endl;
        printf("Res[%d]: %d %d %d %f\n",i,vecRes[i].second->x,vecRes[i].second->y,vecRes[i].second->angle,vecRes[i].first);
    }

    // for(int i=0; i<maxN && pq.size() > 0; i++){
    //     std::pair<float, container*> p = pq.top();
    //     pq.pop();
    //     printf("Res[%d]: %d %d %d %f\n",i,p.second->x,p.second->y,p.second->angle,p.first);
    // }

    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Computation Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - mid).count() << " ms" << std::endl;
    std::cout << "Total Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    // hipFree(d_dataImg);
    // hipFree(d_queryImg);
    // hipFree(d_result);
    // delete(dataImg);
    // delete(result);
    // delete(queryImg);
}

  
// #include <bits/stdc++.h>
// #include <chrono>
// // using namespace std;

// void readImage(int* &img, int &m, int &n, std::string fileName){

//     std::ifstream file(fileName);
//     file >> m >> n;

//     img = new int[m*n*3];

//     for (int i = 0; i < m; i++){
//         for (int j = 0; j < n; j++){
//             int sum = 0;
//             for (int k = 0; k < 3; k++){
//                 file>>img[(i*n+j)*3+k];
//                 sum+=img[(i*n+j)*3+k];
//             }
//         }
//     }
//     file.close();
// }

// __device__
// int d_floor(float x){
//     return floor(x);
// }

// __device__
// int d_ceil(float x){
//     return ceil(x);
// }

// __device__
// int d_round(float x){
//     return llrint(x);
// }
// __device__
// float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){

//     float xx = a - i*cos(theta) - j*sin(theta);
//     float yy = b - i*sin(theta) + j*cos(theta);
//     float x = xx - d_floor(xx);
//     float y = yy - d_floor(yy);
//     if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
//         return 0;
//     }
//     int z00 = dataImg[((d_floor(xx))*N + (d_floor(yy)))*3 + ind];
//     int z01 = dataImg[((d_floor(xx))*N + (d_ceil(yy)))*3 + ind];
//     int z10 = dataImg[((d_ceil(xx))*N + (d_floor(yy)))*3 + ind];
//     int z11 = dataImg[((d_ceil(xx))*N + (d_ceil(yy)))*3 + ind];
//     float cx = 1-x;
//     // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
//     return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
// }

// __global__
// void checkGeneral(int * dataImg, int * queryImg, float * prefix, int M, int N, int m, int n, float queryAvg, double th1, double th2, float pi, float* result){

//     int a,b;
//     int absi = blockIdx.x*256 + threadIdx.x;
//     a = absi/N;
//     b = absi%N;

//     int angles[3] = {45,0,-45};

//     if(a >= M or a < 0 )return ;
//     // printf("At start bid: %d tid: %d\n", blockIdx.x, threadIdx.x);
//     // if(absi > 20000)
//     // printf("abs: %d a: %d b: %d\n",absi, a, b);

//     for(int t=0; t<3; t++){
//         // float sum = 0;    
//         // float theta = angles[t]*pi/180;

//         // // printf("Before interpol bid: %d tid: %d\n", blockIdx.x, threadIdx.x);

//         // for(int i =0; i<m; i++){
//         //     for(int j=0; j<n; j++){
//         //         sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
//         //     }
//         // }

        
//         float theta = angles[t]*pi/180;

//         // if(a == 49 and b == 49)
//         // printf("before a: %d b: %d theta: %f th2: %f, queryAvg: %f\n", a, b, theta,th2,queryAvg);

//         int a1,b1,a2,b2;

//         if(theta < 0){
//             a1 = a + m*sin(theta);
//             b2 = b + m*cos(theta) - n*sin(theta);
//             a2 = a + n*cos(theta);
//             b1 = b;
//         }

//         else{
//             a2 = a;
//             b1 = b - m*sin(theta);
//             b2 = b + n*cos(theta);
//             a1 = a - n*cos(theta) - m*sin(theta);
//         }

//         int denom = abs((a2-a1)*(b2-b1));

//         a1 = max(min(a1,M-1),0);
//         b2 = max(min(b2,N-1),0);
//         b1 = max(min(b1,N-1),0);
//         a2 = max(min(a2,M-1),0);
    
//         float sum = (prefix[a1*N + b1] + prefix[a2*N + b2] - prefix[a2*N + b1] - prefix[a1*N + b2])/denom;

//         // //printf("p: %d q: %d r: %d , s: %d \n", p, q, r, s);
//         // //printf("a1: %d b1: %d a2: %d b2: %d  P: %d Q: %d R: %d S: %d theta: %f \n ", a1, b1,a2,b2 ,a1*N+b1, a2*N+b2, a1*N+b2, a2*N+b1 ,theta);
//         // printf("a1: %d b1: %d a2: %d b2: %d  P: %d Q: %d R: %d S: %d theta: %f \n ", a1, b1,a2,b2 ,p,q,r,s,theta);
//         // //printf("a: %d b: %d val: %d\n", a, b, abs(queryAvg-sum));
//         // //int sum = prefixSum[a1*N + b1] + prefixSum[a2*N + b2] - prefixSum[a1*N + b2] - prefixSum[a2*N + b1];


//         if(abs(queryAvg-sum)<=th2){
//             double sum = 0;
//             for (int i = 0; i<m; i++){
//                 for (int j = 0; j<n; j++){
//                     for (int r = 0; r < 3; r++){
//                         sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[((m-i-1)*n+j)*3+r],2)/(m*n*3);
//                     }
//                 }
//             }
//             // cout << "   -> " <<sqrt(sum) << endl;
//             // printf("    -> %f\n",sqrt(sum));
//             float sq = sqrt(sum);
//             if(sq<=th1){
//                 int ansx =  M- a-1 ;
//                 int ansy =  b ;
//                 // printf("IRes: %d %d %d %f\n",ansx,ansy,t,sq);
//                 result[ansx*N*3 + ansy*3 + t] = sq;
//                 // result[(ansx*N + ansy)*3 = ansx;
//                 return;
//             }
//         }
//     }
// }

// float getAvg(int* &queryImg, int m, int n){
//     float queryAvg = 0;

//     for(int i=0; i<m; i++){        
//         for(int j=0; j<n; j++){
//             float sum = 0;
//             for(int k=0; k<3; k++)
//                 sum += queryImg[(i*n+j)*3+k];
//             queryAvg += sum/3;
//         }
//     }
//     return queryAvg/(m*n);
// }

// class container{

//     public:
//     int x,y,angle;

//     container(int a,int b,int c){
//         x = a;
//         y = b;
//         angle = c;
//     }
// };

// void setSum(int *a, float *&sum, int m, int n){

//     sum = new float[m*n];
//     sum[0] = (a[0]+a[1]+a[2])/3;
 
//     for (int i=1; i<n; i++){
//         float s = 0;
//         for(int k=0; k<3; k++)s+=a[i*3+k];
//         sum[i] = sum[i-1] + s/3; 
//     }
//     for (int i=1; i<m; i++){
//         float s = 0;
//         for(int k=0; k<3; k++)s+=a[i*n*3+k];
//         sum[i*n] = sum[(i-1)*n] + s/3;
//     }
 
//     for (int i=1; i<m; i++){
//         for (int j=1; j<n; j++){
//             float s = 0;
//             for(int k=0; k<3; k++)s+=a[i*n*3+j*3+k];
//             sum[i*n+j] = sum[(i - 1)*n+j] + sum[i*n + j - 1] - sum[(i - 1)*n + j - 1] + s/3;
//         }
//     }
// }


// int main(int argc, char** argv)
// {
//     auto start = std::chrono::high_resolution_clock::now();

//     std::string dataImgPath = argv[1];
//     std::string queryImgPath = argv[2];
//     double th1 = std::stod(argv[3]);
//     double th2 = std::stod(argv[4]);
//     int maxN = std::stoi(argv[5]);

//     int M,N,m,n;

//     int *dataImg;
//     int *queryImg;

//     readImage(dataImg,M,N,dataImgPath);
//     readImage(queryImg,m,n,queryImgPath);
    
//     int *d_dataImg;
//     int *d_queryImg;
     
//     hipMalloc(&d_dataImg, (M*N*3)*sizeof(int));
//     hipMalloc(&d_queryImg, (m*n*3)*sizeof(int));

//     hipMemcpy(d_dataImg, dataImg, (M*N*3)*sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_queryImg, queryImg, (m*n*3)*sizeof(int), hipMemcpyHostToDevice);

//     float *dataPrefix;
//     float *d_dataPrefix;
//     setSum(dataImg,dataPrefix,M,N);
//     hipMalloc(&d_dataPrefix, (M*N)*sizeof(float));
//     hipMemcpy(d_dataPrefix, dataPrefix, (M*N)*sizeof(float), hipMemcpyHostToDevice);

//     float *result = new float[N*M*3];
//     for(int i=0; i<M; i++){
//         for(int j=0; j<N; j++){
//             for(int k=0; k<3; k++){
//                 result[i*N*3+j*3+k]=-1;
//             }
//         }
//     }
//     float *d_result;
//     hipMalloc(&d_result, M*N*3*sizeof(float));
//     hipMemcpy(d_result, result, M*N*3*sizeof(float), hipMemcpyHostToDevice);

//     float queryAvg = getAvg(queryImg, m,n);
//     // checkGeneral(dataImg, queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180);

//     auto mid = std::chrono::high_resolution_clock::now();

//     std::cout << "Pre processing Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(mid - start).count() << " ms" << std::endl;

//     checkGeneral<<<(N*M+255)/256, 256>>>(d_dataImg, d_queryImg, d_dataPrefix, M,N,m,n,queryAvg,th1,th2,M_PI,d_result);

//     hipMemcpy(result, d_result, M*N*3*sizeof(float), hipMemcpyDeviceToHost);

//     std::priority_queue <std::pair<float, container*> > pq;
//     int angles[3] = {45,0,-45};
    
//     for(int i=0; i<M; i++){
//         for(int j=0; j<N; j++){
//             for(int k=0; k<3; k++){
//                 if(result[i*N*3+j*3+k]!=-1){
//                     // printf("%d %d %d %f\n",i,j,k,result[i*N*3+j*3+k]);
//                     container* c = new container(i,j,angles[k]);
//                     pq.push({result[i*N*3+j*3+k], c});
//                 }
//                 if(pq.size()>maxN)pq.pop();
//             }
//         }
//     }
//     //std::cout<<"pq size: "<<pq.size()<<std::endl;
//     int pqSize = pq.size();
//     std::vector<std::pair<float, container*>> vecRes;
//     for(int i = 0;i<pqSize;i++){
//         vecRes.push_back(pq.top());
//         pq.pop();
//     }
//     std::ofstream outfile("output.txt");
//     std::cout << std::endl;
//     reverse(vecRes.begin(), vecRes.end());
//     for(int i = 0;i<vecRes.size();i++){
//         outfile << vecRes[i].second->x << " " << vecRes[i].second->y << " " << vecRes[i].second->angle << std::endl;
//         // printf("Res[%d]: %d %d %d %f\n",i,vecRes[i].second->x,vecRes[i].second->y,vecRes[i].second->angle,vecRes[i].first);
//     }
//     // for(int i=0; i<maxN && pq.size() > 0; i++){
//     //     std::pair<float, container*> p = pq.top();
//     //     pq.pop();
//     //     printf("Res[%d]: %d %d %d %f\n",i,p.second->x,p.second->y,p.second->angle,p.first);
//     // }

//     auto end = std::chrono::high_resolution_clock::now();
//     std::cout << "Computation Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - mid).count() << " ms" << std::endl;
//     std::cout << "Total Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

//     std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

//     // hipFree(d_dataImg);
//     // hipFree(d_queryImg);
//     // hipFree(d_result);
//     // delete(dataImg);
//     // delete(result);
//     // delete(queryImg);
// }