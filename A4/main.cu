
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void readImage(int*** &img, int &m, int &n, string fileName){

    ifstream file(fileName);
    file >> m >> n;

    img = new int**[m];

    for (int i = 0; i < m; i++)
    {
        img[i] = new int*[n];
        for (int j = 0; j < n; j++)
        {
            img[i][j] = new int[4];
            int sum = 0;
            for (int k = 0; k < 3; k++)
            {
                file>>img[i][j][k];
                sum+=img[i][j][k];
            }
            img[i][j][3] = sum/3;   
        }
    }
    file.close();
}



bool RMSD(int*** &dataImg,int*** &queryImg,int i,int j,int M,int N,int m,int n,double th1){
    double sum = 0;
    for (int p = i; p < i+m; p++)
    {
        for (int q = j; q < j+n; q++)
        {
            for (int r = 0; r < 3; r++)
            {
                sum+=pow(dataImg[p][q][r]-queryImg[p-i][q-j][r],2)/(m*n*3);
            }
            
        }
        
    }
    cout<<i<<" " <<j<<" "<<sqrt(sum)<<" "<<th1<<endl;
    if(sqrt(sum)<=th1){
        return true;
    }
    return false;
    
}
pair<int,int> templateSearchBasic(int*** &dataImg,int*** &queryImg,int** &dataImgAvg,int M,int N,int m,int n,double th1,double th2,double queryAvg){
    for (int i = 0; i <= M-m; i++)
    {
        for (int j = 0; j <= N-n; j++)
        {
            if(abs(queryAvg-dataImgAvg[i][j])<=th2){      
                //cout<<"hi "<<endl; 
                if(RMSD(dataImg,queryImg,i,j,M,N,m,n,th1)){
                    return make_pair(i,j);
                }
            }
        }
        
    }
    return make_pair(-1,-1);
}


template <typename T>
void initialize2Darray(T** &arr,int m,int n) {
    arr = new T*[m];
    for (int i = 0; i < m; i++)
    {
        arr[i] = new T[n];
    }
}

void checkZero(int *** &dataImg, int *** &queryImg, int M, int N, int m, int n, int queryAvg, double th1, double th2){
    int **dataImgTotalSum,**dataImgAvg;
    initialize2Darray<int>(dataImgTotalSum,M,N);
    initialize2Darray<int>(dataImgAvg,M,N);

    for (int i = 0; i <= M - m; i++)
    {
        for (int j = 0; j <= N - n; j++)
        {
            dataImgTotalSum[i][j] = 0;
            if(i == 0){
                if(j == 0){
                    for(int a = 0; a < m; a++){
                        for(int b = 0; b < n; b++){
                            dataImgTotalSum[i][j]+=dataImg[i+a][j+b][3];
                        }
                    }
                }
                else{
                    dataImgTotalSum[i][j] = dataImgTotalSum[i][j-1];
                    for(int a=0; a<m; a++){
                        dataImgTotalSum[i][j]+= dataImg[i+a][j+n-1][3] - dataImg[i+a][j-1][3];
                    }
                }
            }
            else{
                dataImgTotalSum[i][j] = dataImgTotalSum[i-1][j];
                for(int a=0; a<n; a++){
                    dataImgTotalSum[i][j]+= dataImg[i+m-1][j+a][3] - dataImg[i-1][j+a][3];
                }
            }
        }
    }
    pair<int,int> pos = templateSearchBasic(dataImg,queryImg,dataImgTotalSum,M,N,m,n,th1,th2,queryAvg);
    cout<<"Res: "<<pos.first<<" "<<pos.second<<endl;
}

float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int*** &dataImg, int ind){
    float xx = a + i*cos(theta) - j*sin(theta);
    float yy = b + i*sin(theta) + j*cos(theta);
    float x = xx - floor(xx);
    float y = yy - floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[(int)floor(xx)][(int)floor(yy)][ind];
    int z01 = dataImg[(int)floor(xx)][(int)ceil(yy)][ind];
    int z10 = dataImg[(int)ceil(xx)][(int)floor(yy)][ind];
    int z11 = dataImg[(int)ceil(xx)][(int)ceil(yy)][ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

void checkGeneral(int *** &dataImg, int *** &queryImg, int M, int N, int m, int n, int queryAvg, double th1, double th2, float theta){

    for(int a=48; a<52; a++){
        for(int b=48; b<52;b ++){
            float sum = 0;
            for(int i =0; i<m; i++){
                for(int j=0; j<n; j++){
                    // cout << sum << endl;
                    sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
                }
            }
            cout << "a: " << a << " b: " << b << " " << abs(queryAvg-sum)/(m*n) << endl;
            if(abs(queryAvg-sum)<=th2){
                double sum = 0;
                for (int i = 0; i<m; i++){
                    for (int j = 0; j<n; j++){
                        for (int r = 0; r < 3; r++){
                            sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[i][j][r],2)/(m*n*3);
                        }
                    }
                }
                cout << "   -> " <<sqrt(sum) << endl;
                if(sqrt(sum)<=th1){
                    cout<<"Res: "<<a<<" "<<b<< " " << (int)(theta*180/M_PI) << endl;
                    return;
                }
            }
        }
    }
}



int main(int argc, char** argv)
{
    string dataImgPath = argv[1];
    string queryImgPath = argv[2];
    double th1 = stod(argv[3]);
    double th2 = stod(argv[4]);
    int maxN = stoi(argv[5]);

    int M,N,m,n;

    int ***dataImg;
    int ***queryImg;
    
    readImage(dataImg,M,N,dataImgPath);
    readImage(queryImg,m,n,queryImgPath);

    th2*=m*n;
           

    int queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            queryAvg+=queryImg[i][j][3];
        }
    }
    
    int ***d_dataImg,**d_queryImg;
    //float *x, *y, *d_x, *d_y;
    //x = (float*)malloc(N*sizeof(float));
    //y = (float*)malloc(N*sizeof(float));

    //cudaMalloc(&d_x, N*sizeof(float)); 
    //cudaMalloc(&d_y, N*sizeof(float));
    hipMalloc((void **) &d_dataImg, (M*N*4)*sizeof(int));
    hipMalloc((void **) &d_queryImg, (m*n*4)*sizeof(int));


    // for (int i = 0; i < N; i++) {
    //     x[i] = 1.0f;
    //     y[i] = 2.0f;
    // }

    hipMemcpy(d_dataImg, dataImg, (M*N*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*4)*sizeof(int), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    // saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    // cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //     maxError = max(maxError, abs(y[i]-4.0f));
    // printf("Max error: %f\n", maxError);

    hipFree(d_dataImg);
    hipFree(d_queryImg);
    // free(x);
    // free(y);
}