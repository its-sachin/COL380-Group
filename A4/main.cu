
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void readImage(int* &img, int &m, int &n, string fileName){

    ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*4];

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            int sum = 0;
            for (int k = 0; k < 3; k++)
            {
                file>>img[(i*n+j)*4+k];
                sum+=img[(i*n+j)*4+k];
            }
            img[(i*n+j)*4+3] = sum/3;   
        }
    }
    file.close();
}


float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){
    float xx = a + i*cos(theta) - j*sin(theta);
    float yy = b + i*sin(theta) + j*cos(theta);
    float x = xx - floor(xx);
    float y = yy - floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[(((int)floor(xx))*N + ((int)floor(yy)))*4 + ind];
    int z01 = dataImg[(((int)floor(xx))*N + ((int)ceil(yy)))*4 + ind];
    int z10 = dataImg[(((int)ceil(xx))*N + ((int)floor(yy)))*4 + ind];
    int z11 = dataImg[(((int)ceil(xx))*N + ((int)ceil(yy)))*4 + ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

void checkGeneral(int * &dataImg, int * &queryImg, int M, int N, int m, int n, int queryAvg, double th1, double th2, float theta){

    for(int a=48; a<52; a++){
        for(int b=48; b<52;b ++){
            float sum = 0;
            for(int i =0; i<m; i++){
                for(int j=0; j<n; j++){
                    // cout << sum << endl;
                    sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
                }
            }
            cout << "a: " << a << " b: " << b << " " << abs(queryAvg-sum)/(m*n) << endl;
            if(abs(queryAvg-sum)<=th2){
                double sum = 0;
                for (int i = 0; i<m; i++){
                    for (int j = 0; j<n; j++){
                        for (int r = 0; r < 3; r++){
                            sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[(i*n+j)*4+r],2)/(m*n*3);
                        }
                    }
                }
                cout << "   -> " <<sqrt(sum) << endl;
                if(sqrt(sum)<=th1){
                    cout<<"Res: "<<M-round(a + m*cos(theta) )<<" "<<round(b + n*sin(theta) )<< " " << (int)(theta*180/M_PI) << endl;
                    return;
                }
            }
        }
    }
}



int main(int argc, char** argv)
{
    string dataImgPath = argv[1];
    string queryImgPath = argv[2];
    double th1 = stod(argv[3]);
    double th2 = stod(argv[4]);
    int maxN = stoi(argv[5]);

    int M,N,m,n;

    int *dataImg;
    int *queryImg;
    
    readImage(dataImg,M,N,dataImgPath);
    readImage(queryImg,m,n,queryImgPath);

    th2*=m*n;
           

    int queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            queryAvg+=queryImg[(i*n+j)*4+3];
        }
    }
    
    int *d_dataImg;
    int* d_queryImg;
   
    hipMalloc(&d_dataImg, (M*N*4)*sizeof(int));
    hipMalloc(&d_queryImg, (m*n*4)*sizeof(int));

    hipMemcpy(d_dataImg, dataImg, (M*N*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*4)*sizeof(int), hipMemcpyHostToDevice);

    hipFree(d_dataImg);
    hipFree(d_queryImg);
    free(dataImg);
    free(queryImg);
}