
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <chrono>
// using namespace std;

void readImage(int* &img, int &m, int &n, std::string fileName, long long *&prefixSum, bool isData = false){

    std::ifstream file(fileName);
    file >> m >> n;

    img = new int[m*n*4];

    if(isData){
        prefixSum = new long long[m*n];
    }

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            int sum = 0;
            for (int k = 0; k < 3; k++){
                file>>img[(i*n+j)*4+k];
                sum+=img[(i*n+j)*4+k];
            }
            img[(i*n+j)*4+3] = sum/3;  
            if(isData){
                prefixSum[i*n+j] = sum/3;
                if(i == 0){
                    if(j!=0){
                        prefixSum[j] += prefixSum[j-1];
                    }
                }
                else{
                    if(j==0)
                        prefixSum[i*n+j] += prefixSum[(i-1)*n];
                    else
                        prefixSum[i*n+j] += prefixSum[(i-1)*n+j] + prefixSum[i*n + j-1] - prefixSum[(i-1)*n+j-1];
                }
            } 
        }
    }
    file.close();
}

__device__
int d_floor(float x){
    return floor(x);
}

__device__
int d_ceil(float x){
    return ceil(x);
}

__device__
int d_round(float x){
    return llrint(x);
}

__device__
float getInterpolated(int a, int b, int i, int j, float theta, int M, int N, int* &dataImg, int ind){
    float xx = a + i*cos(theta) - j*sin(theta);
    float yy = b + i*sin(theta) + j*cos(theta);
    float x = xx - d_floor(xx);
    float y = yy - d_floor(yy);
    if(xx<0 || ceil(xx)>=M || yy<0 || ceil(yy)>=N){
        return 0;
    }
    int z00 = dataImg[((d_floor(xx))*N + (d_floor(yy)))*4 + ind];
    int z01 = dataImg[((d_floor(xx))*N + (d_ceil(yy)))*4 + ind];
    int z10 = dataImg[((d_ceil(xx))*N + (d_floor(yy)))*4 + ind];
    int z11 = dataImg[((d_ceil(xx))*N + (d_ceil(yy)))*4 + ind];
    float cx = 1-x;
    // cout << "x: " << x << " y: " << y << " cx: " << cx << " z01: "<< z00 << " " <<z01 << " " <<z10 << " " <<z11<< endl;
    return ( (z00*cx + z10*x)*(1-y) + (z01*cx + z11*x)*y );
}

__global__
void checkGeneral(int * dataImg, int * queryImg, long long * prefixSum, int M, int N, int m, int n, int queryAvg, double th1, double th2, float pi, float* result){

    // printf("%d\n", prefixSum[200]);
    // printf("M: %d N:%d",M,N);
    // for(int i = 0;i<M*N;i++){
    //     printf("i:%d  \n",i);
    //     //printf("i:%d %d \n",i, prefixSum[i]);
    // }
    // return;
    
    int a,b;
    int absi = blockIdx.x*256 + threadIdx.x;
    a = absi/N;
    b = absi%N;

    //printf("On a b %d %d %d \n",a,b,prefixSum[a*N+b]);

    int angles[3] = {45,0,-45};
    // printf("At start bid: %d tid: %d\n", blockIdx.x, threadIdx.x);
    // if(absi > 20000)
    // printf("abs: %d a: %d b: %d\n",absi, a, b);

    for(int t=0; t<3; t++){
        // float sum = 0;    
        // float theta = angles[t]*pi/180;

        // // printf("Before interpol bid: %d tid: %d\n", blockIdx.x, threadIdx.x);

        // for(int i =0; i<m; i++){
        //     for(int j=0; j<n; j++){
        //         sum += getInterpolated(a,b,i,j,theta,M,N,dataImg,3);
        //     }
        // }

        //printf("a: %d b: %d val: %d\n", a, b, abs(queryAvg-sum));

        float theta = angles[t]*pi/180;

        int a1 = a - n*sin(theta);
        int b1 = b;
        int b2 = b + n*cos(theta) + m*sin(theta);
        int a2 = a + m*cos(theta);
        long long p,q,r,s;
        if(a1*N+b1<0 || a1*N+b1>=M*N){
            p = 0;
        }else{
            p = prefixSum[a1*N+b1];
        }
        if(a2*N+b2<0 || a2*N+b2>=M*N){
            q = 0;
        }else{
            q = prefixSum[a2*N+b2];
        }
        if(a1*N+b2<0 || a1*N+b2>=M*N){
            r = 0;
        }else{
            r = prefixSum[a1*N+b2];
        }
        if(a2*N+b1<0 || a2*N+b1>=M*N){
            s = 0;
        }else{
            s = prefixSum[a2*N+b1];
        }
        // for(int i = a1;i<a2;i++){
        //     for(int j=b1; j<b2; j++){
        //         printf("(At i:%d j:%d prefixSum: %d )",i,j, prefixSum[i*N+j]);
        //     }
        // }
        // printf("\n");

        // long long sum = (p + q) - (r + s);
        long long sum = (p -r) + (q - s);
        //printf("a: %d \n", sum);
        //printf("p: %d q: %d r: %d , s: %d , a1*N+b1: %d , a2*N+b2: %d , a1*N+b2: %d ,a2*N+b1: %d  \n", p, q, r, s, a1*N+b1, a2*N+b2, a1*N+b2, a2*N+b1);
        //printf("a1: %d b1: %d a2: %d b2: %d  P: %d Q: %d R: %d S: %d theta: a:%d b:%d n:%d m*sin(theta)%f \n ", a1, b1,a2,b2 ,a1*N+b1, a2*N+b2, a1*N+b2, a2*N+b1 ,a,b,n,m*sin(theta));
        //printf("a1: %d b1: %d a2: %d b2: %d  P: %d Q: %d R: %d S: %d theta: %f \n ", a1, b1,a2,b2 ,p,q,r,s,theta);
        //printf("a: %d b: %d val: %d\n", a, b, abs(queryAvg-sum));
        //int sum = prefixSum[a1*N + b1] + prefixSum[a2*N + b2] - prefixSum[a1*N + b2] - prefixSum[a2*N + b1];

        // printf("a: %d b: %d sum: %d , queryAvg: %d , absDiff: %d, th2: %f \n", a, b,sum,queryAvg,abs(queryAvg-sum),th2);

        if(abs(queryAvg-sum)<=th2){
            double sum = 0;
            for (int i = 0; i<m; i++){
                for (int j = 0; j<n; j++){
                    for (int r = 0; r < 3; r++){
                        sum+=pow(getInterpolated(a,b,i,j,theta,M,N,dataImg,r)-queryImg[(i*n+j)*4+r],2)/(m*n*3);
                    }
                }
            }
            // cout << "   -> " <<sqrt(sum) << endl;
            // printf("    -> %f\n",sqrt(sum));
            float sq = sqrt(sum);
            if(sq<=th1){
                int ansx = M-d_round(a + m*cos(theta) );
                int ansy = d_round(b + m*sin(theta) );
                // printf("IRes: %d %d %d %f\n",ansx,ansy,t,sq);
                result[ansx*N*3 + ansy*3 + t] = sq;
                // result[(ansx*N + ansy)*3 = ansx;
                return;
            }
        }
    }
}

int getAvg(int* &queryImg, int m, int n){
    int queryAvg = 0;

    for(int i=0; i<m; i++){        
        for(int j=0; j<n; j++){
            queryAvg+=queryImg[(i*n+j)*4+3];
        }
    }
    return queryAvg;
}

class container{

    public:
    int x,y,angle;

    container(int a,int b,int c){
        x = a;
        y = b;
        angle = c;
    }
};


int main(int argc, char** argv)
{
    auto start = std::chrono::high_resolution_clock::now();

    std::string dataImgPath = argv[1];
    std::string queryImgPath = argv[2];
    double th1 = std::stod(argv[3]);
    double th2 = std::stod(argv[4]);
    int maxN = std::stoi(argv[5]);

    int M,N,m,n;

    int *dataImg;
    int *queryImg;
    long long *dataPrefix;

    readImage(dataImg,M,N,dataImgPath,dataPrefix,true);
    readImage(queryImg,m,n,queryImgPath,dataPrefix);

    // std::cout<<"hello \n";
    // for(int i = 0;i<M*N;i++){
    //     std::cout<<dataPrefix[i]<<" " <<i<<"\n";
    // }

    th2*=m*n;
    
    int *d_dataImg;
    int *d_queryImg;
    long long *d_dataPrefix;
    
   
    hipMalloc(&d_dataImg, (M*N*4)*sizeof(int));
    hipMalloc(&d_queryImg, (m*n*4)*sizeof(int));
    hipMalloc(&d_dataPrefix, (M*N)*sizeof(long long));

    hipMemcpy(d_dataImg, dataImg, (M*N*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryImg, queryImg, (m*n*4)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dataPrefix, dataPrefix, (M*N)*sizeof(long long), hipMemcpyHostToDevice);

    float *result = new float[N*M*3];
    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<3; k++){
                result[i*N*3+j*3+k]=-1;
            }
        }
    }
    float *d_result;
    hipMalloc(&d_result, M*N*3*sizeof(float));
    hipMemcpy(d_result, result, M*N*3*sizeof(float), hipMemcpyHostToDevice);

    int queryAvg = getAvg(queryImg, m,n);
    // checkGeneral(dataImg, queryImg, M,N,m,n,queryAvg,th1,th2,45*M_PI/180);

    auto mid = std::chrono::high_resolution_clock::now();

    std::cout << "Pre processing Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(mid - start).count() << " ms" << std::endl;

    //std::cout << d_dataPrefix[0]<<" \n";
    checkGeneral<<<(N*M+255)/256, 256>>>(d_dataImg, d_queryImg, d_dataPrefix, M,N,m,n,queryAvg,th1,th2,M_PI,d_result);
    //checkGeneral<<<1,1>>>(d_dataImg, d_queryImg, d_dataPrefix, M,N,m,n,queryAvg,th1,th2,M_PI,d_result);

    hipMemcpy(result, d_result, M*N*3*sizeof(float), hipMemcpyDeviceToHost);

    std::priority_queue <std::pair<float, container*> > pq;
    int angles[3] = {45,0,-45};
    
    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<3; k++){
                if(result[i*N*3+j*3+k]!=-1){
                    printf("%d %d %d %f\n",i,j,k,result[i*N*3+j*3+k]);
                    container* c = new container(i,j,angles[k]);
                    pq.push({result[i*N*3+j*3+k], c});
                }
            }
        }
    }

    std::cout << std::endl;
    for(int i=0; i<maxN && pq.size() > 0; i++){
        std::pair<float, container*> p = pq.top();
        pq.pop();
        printf("Res[%d]: %d %d %d %f\n",i,p.second->x,p.second->y,p.second->angle,p.first);
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Computation Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - mid).count() << " ms" << std::endl;
    std::cout << "Total Time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    hipFree(d_dataImg);
    hipFree(d_queryImg);
    hipFree(d_result);
    delete(dataImg);
    delete(result);
    delete(queryImg);
}
